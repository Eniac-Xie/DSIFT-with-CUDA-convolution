#include <iostream>
#include <fstream>
#include <windows.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include ""
#include "hip/device_functions.h"
#include "DSIFT_CUDA.cuh"

using namespace std;
using namespace cv;


int main(int argc, char** argv)
{
	IplImage *srcImage = 0;
	char *srcPictureName = "data/2.jpg";
	int imageWidth = 0, imageHeight = 0;

	/* parameter of descripter */
	int numBinX = 4, numBinY = 4, numBinT = 8, binSizeX = 8, binSizeY = 8;

	/* step of X and Y */
	int step[2] = { 1, 1 };

	srcImage = cvLoadImage(srcPictureName, 1);
	imageWidth = srcImage->width;
	imageHeight = srcImage->height;

	if (srcImage == NULL)
	{
		cout << "Image not found" << endl;
		exit(1);
	}
	IplImage * grayImage = cvCreateImage(cvSize(srcImage->width, srcImage->height), srcImage->depth, 1);
	cvCvtColor(srcImage, grayImage, CV_BGR2GRAY);

	/*initialize Dsift Filter*/

	LARGE_INTEGER t1, t3, tc;
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);

	DsiftDescriptorGeometry* geom = init_dsift_geom(numBinX, numBinY, numBinT, binSizeX, binSizeY);
	DsiftFilter* self = init_dsift_filter(imageWidth, imageHeight, geom, step);
	dsift_alloc_buffers(self);
	float *srcGPU, *destGPU;
	checkCudaErrors(hipMalloc(&srcGPU, sizeof(float)* self->numFrames * self->descrSize));
	checkCudaErrors(hipMalloc(&destGPU, sizeof(float)* self->numFrames * self->descrSize));
	compute_grad(self, grayImage);
	
	dsift_with_gaussian_window(self, srcGPU);

	DsiftKeypoint* frameIter = self->frames;

	dim3 threads(128, 4);
	dim3 blocks(64, 64, self->numFrames / 16384);
	reverse << <blocks, threads >> >(srcGPU, destGPU, self->numFrames, self->descrSize);
	checkCudaErrors(hipMemcpy(self->descrs, destGPU, sizeof(float)* self->numFrames * self->descrSize, hipMemcpyDeviceToHost));

	int framex, framey, bint;
	int frameSizeX = self->geom.binSizeX * (self->geom.numBinX - 1) + 1;
	int frameSizeY = self->geom.binSizeY * (self->geom.numBinY - 1) + 1;
	int descrSize = dsift_get_descriptor_size(self);
	float * descrIter = self->descrs;
	float deltaCenterX = 0.5F * self->geom.binSizeX * (self->geom.numBinX - 1);
	float deltaCenterY = 0.5F * self->geom.binSizeY * (self->geom.numBinY - 1);

	for (framey = self->boundMinY;
		framey <= self->boundMaxY - frameSizeY + 1;
		framey += self->stepY) 
	{
		for (framex = self->boundMinX;
			framex <= self->boundMaxX - frameSizeX + 1;
			framex += self->stepX) 
		{
			frameIter->x = framex + deltaCenterX;
			frameIter->y = framey + deltaCenterY;
			frameIter++;
		} /* for framex */
	} /* for framey */

#pragma omp parallel for
	for (int i = 0; i < self->numFrames; i++)
	{
		/* L2 normalize */
		dsift_normalize_histogram(descrIter, descrIter + descrSize);

		/* clamp */
		for (bint = 0; bint < descrSize; ++bint)
		if (descrIter[bint] > 0.2F) descrIter[bint] = 0.2F;

		/* L2 normalize */
		dsift_normalize_histogram(descrIter, descrIter + descrSize);

		descrIter += descrSize;
	}

	QueryPerformanceCounter(&t3);
	printf("Use Time:%f\n", (t3.QuadPart - t1.QuadPart)*1.0 / tc.QuadPart);
	ofstream out("output.txt");
	DsiftKeypoint const *frames = self->frames;
	ofstream outFrames("Frames.txt");
	for (int i = 0; i < self->numFrames; i++)
	{
		outFrames << frames[i].y << "\t" << frames[i].x << "\t";
		outFrames << endl;
		float *tmpDescr = self->descrs + descrSize * i;
		for (int j = 0; j < descrSize; ++j) 
		{
			unsigned char res = (unsigned char)(512.0F * tmpDescr[j] < 255.0F ? (512.0F * tmpDescr[j]) : 255.0F);
			out << (unsigned int)res << "\t";
		}
		out << endl;
	}
	out.close();
	outFrames.close();
	cvNamedWindow("srcImage", 0);
	cvShowImage("srcImage", srcImage);
	cvWaitKey(0);
	return 0;
}